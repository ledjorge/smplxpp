#include "hip/hip_runtime.h"
#include <iostream>

#include "smplx/smplx.hpp"
#include "smplx/util.hpp"
#include "smplx/internal/cuda_util.cuh"

namespace smplx {
namespace {
using cuda_util::device::BLOCK_SIZE;
using cuda_util::from_host_eigen_sparse_matrix;
using cuda_util::from_host_eigen_matrix;
using cuda_util::to_host_eigen_matrix;

namespace device {
/** Rodrigues formula: d_pose_full (#joints,3) -> out(#joints,9);
 * also copies out to upper-left 3x3 part of out_joint_local_transform
 * (#joints,12)
 * Note this is no longer used; however I have not deleted it since
 * it may be useful in the future */
/* __global__ void rodrigues(float* RESTRICT d_pose_full, float* RESTRICT out,
                          float* RESTRICT out_joint_local_transform) {
    const int in_idx = threadIdx.x * 3;
    const int out_idx = threadIdx.x * 9;
    const int out_transform_idx = threadIdx.x * 12;
    float theta = norm3df(d_pose_full[in_idx], d_pose_full[in_idx + 1],
                          d_pose_full[in_idx + 2]);
    if (fabsf(theta) < 1e-5f) {
        for (int i = out_idx; i < out_idx + 9; ++i) {
            out[i] = 0.f;
        }
        out_joint_local_transform[out_transform_idx + 1] =
            out_joint_local_transform[out_transform_idx + 2] =
                out_joint_local_transform[out_transform_idx + 4] =
                    out_joint_local_transform[out_transform_idx + 6] =
                        out_joint_local_transform[out_transform_idx + 8] =
                            out_joint_local_transform[out_transform_idx + 9] =
                                0.f;
        out_joint_local_transform[out_transform_idx] =
            out_joint_local_transform[out_transform_idx + 5] =
                out_joint_local_transform[out_transform_idx + 10] = 1.f;
    } else {
        float cm1 = cos(theta) - 1.f;
        float s = sin(theta);

        const float a = d_pose_full[in_idx] /= theta;
        const float b = d_pose_full[in_idx + 1] /= theta;
        const float c = d_pose_full[in_idx + 2] /= theta;

        out[out_idx] = cm1;
        out[out_idx + 1] = -s * c;
        out[out_idx + 2] = s * b;
        out[out_idx + 3] = s * c;
        out[out_idx + 4] = cm1;
        out[out_idx + 5] = -s * a;
        out[out_idx + 6] = -s * b;
        out[out_idx + 7] = s * a;
        out[out_idx + 8] = cm1;

        for (int j = 0; j < 3; ++j) {
            for (int k = 0; k < 3; ++k) {
                out_joint_local_transform[out_transform_idx + j * 4 + k] =
                    (out[out_idx + j * 3 + k] -=
                     cm1 * d_pose_full[in_idx + j] * d_pose_full[in_idx + k]);
            }
            // Un-subtract identity
            out_joint_local_transform[out_transform_idx + j * 4 + j] += 1.f;
        }
    }
} */

/** Joint regressor: multiples sparse matrix in CSR represented by
 *  (model_jr_values(nnz), ..inner(nnz), ..outer(#joints+1)) to
 *  d_verts_shaped(#verts,3) row-major
 *  -> outputs to out(#joints, 3) row-major
 *  TODO: Optimize. The matrix is very wide and this is not efficient */
__global__ void joint_regressor(float* RESTRICT d_verts_shaped, float* RESTRICT model_jr_values,
                                int* RESTRICT model_jr_inner, int* RESTRICT model_jr_outer,
                                float* RESTRICT out_joints) {
    const int joint = threadIdx.y, idx = threadIdx.x;
    out_joints[joint * 3 + idx] = 0.f;
    for (int i = model_jr_outer[joint]; i < model_jr_outer[joint + 1]; ++i) {
        out_joints[joint * 3 + idx] +=
            model_jr_values[i] * d_verts_shaped[model_jr_inner[i] * 3 + idx];
    }
}

/** Linear blend skinning kernel.
  * d_joint_global_transform (#joints, 12) row-major;
  *   global-space homogeneous transforms (bottom row dropped)
  *   at each joint from local_to_global
  * d_points_shaped (#points, 3) row-major; vertices after blendshapes applied
  * (model_weights_values(nnz), ..inner(nnz), ..outer(#joints+1)) sparse LBS weights in CSR
  * -> out_verts(#points, 3) resulting vertices after deformation */
__global__ void lbs(float* RESTRICT d_joint_global_transform, float* RESTRICT d_verts_shaped,
                    float* RESTRICT model_weights_values, int* RESTRICT model_weights_inner,
                    int* RESTRICT model_weights_outer,
                    float* RESTRICT out_verts,  // transformed joint pos
                    const int n_joints, const int n_verts) {
    const int vert = blockDim.x * blockIdx.x + threadIdx.x;  // Vert idx
    if (vert < n_verts) {
        for (int i = 0; i < 3; ++i) {
            out_verts[vert * 3 + i] = 0.f;
            for (int joint_it = model_weights_outer[vert];
                 joint_it < model_weights_outer[vert + 1]; ++joint_it) {
                const int joint_row_idx =
                    model_weights_inner[joint_it] * 12 + i * 4;
                for (int j = 0; j < 3; ++j) {
                    out_verts[vert * 3 + i] +=
                        model_weights_values[joint_it] *
                        d_joint_global_transform[joint_row_idx + j] *
                        d_verts_shaped[vert * 3 + j];
                }
                out_verts[vert * 3 + i] +=
                    model_weights_values[joint_it] *
                    d_joint_global_transform[joint_row_idx + 3];
            }
        }
    }
}

}  // namespace device
}  // namespace

/*
struct {
   float* params = nullptr;
   float* verts = nullptr;
   float* blendshape_params = nullptr;
   float* joint_transforms = nullptr;
} device; */
template<class ModelConfig>
__host__ void Body<ModelConfig>::_cuda_load() {
    cudaCheck(hipMalloc((void**)&device.verts, model.n_verts() * 3 * sizeof(float)));
    cudaCheck(hipMalloc((void**)&device.blendshape_params,
               model.n_blend_shapes() * sizeof(float)));
    cudaCheck(hipMalloc((void**)&device.joint_transforms,
               model.n_joints() * 12 * sizeof(float)));
    cudaCheck(hipMalloc((void**)&device.verts_shaped,
                         model.n_verts() * 3 * sizeof(float)));
    cudaCheck(hipMalloc((void**)&device.joints_shaped,
                         model.n_joints() * 3 * sizeof(float)));
}
template<class ModelConfig>
__host__ void Body<ModelConfig>::_cuda_free() {
    if (device.verts) hipFree(device.verts);
    if (device.blendshape_params) hipFree(device.blendshape_params);
    if (device.joint_transforms) hipFree(device.joint_transforms);
    if (device.verts_shaped) hipFree(device.verts_shaped);
    if (device.joints_shaped) hipFree(device.joints_shaped);
}
template<class ModelConfig>
__host__ void Body<ModelConfig>::_cuda_maybe_retrieve_verts() const {
    if (!_verts_retrieved) {
        _verts.resize(model.n_verts(), 3);
        hipMemcpy(_verts.data(), device.verts, _verts.size() * sizeof(float),
                   hipMemcpyDeviceToHost);
        _verts_retrieved = true;
    }
}

template<class ModelConfig>
__host__ void Body<ModelConfig>::_cuda_maybe_retrieve_verts_shaped() const {
    if (!_verts_shaped_retrieved) {
        _verts_shaped.resize(model.n_verts(), 3);
        hipMemcpy(_verts_shaped.data(), device.verts_shaped,
                    _verts_shaped.size() * sizeof(float),
                   hipMemcpyDeviceToHost);
        _verts_shaped_retrieved = true;
    }
}


template<class ModelConfig>
SMPLX_HOST void Body<ModelConfig>::_cuda_update(
        float* h_blendshape_params,
        float* h_joint_transforms,
        bool enable_pose_blendshapes) {
    // Verts will be updated
    _verts_retrieved = false;
    _verts_shaped_retrieved = false;

    // Copy parameters to GPU
    cudaCheck(hipMemcpyAsync(device.blendshape_params, h_blendshape_params,
                ModelConfig::n_blend_shapes() * sizeof(float),
               hipMemcpyHostToDevice));
    // Shape blendshapes
    cudaCheck(hipMemcpyAsync(device.verts_shaped, model.device.verts,
               model.n_verts() * 3 * sizeof(float), hipMemcpyDeviceToDevice));
    cuda_util::mmv_block<float, true>(model.device.blend_shapes,
            device.blendshape_params, device.verts_shaped, ModelConfig::n_verts() * 3,
            ModelConfig::n_shape_blends());

    // Joint regressor
    // TODO: optimize sparse matrix multiplication, maybe use ELL format
    dim3 jr_blocks(3, model.n_joints());
    device::joint_regressor<<<1, jr_blocks>>>(
        device.verts_shaped, model.device.joint_reg.values, model.device.joint_reg.inner,
        model.device.joint_reg.outer, device.joints_shaped);

    if (enable_pose_blendshapes) {
        // Pose blendshapes.
        // Note: this is the most expensive operation.
        cuda_util::mmv_block<float, true>(model.device.blend_shapes + ModelConfig::n_shape_blends() * 3 * ModelConfig::n_verts(),
               device.blendshape_params + ModelConfig::n_shape_blends(), device.verts_shaped, ModelConfig::n_verts() * 3,
               ModelConfig::n_pose_blends());
    }

    // Compute global joint transforms, this part can't be parallized and
    // is horribly slow on GPU; we do it on CPU instead
    // Actually, this is pretty bad too, TODO try implementing on GPU again
    cudaCheck(hipMemcpyAsync(_joints_shaped.data(), device.joints_shaped, model.n_joints() * 3 * sizeof(float),
               hipMemcpyDeviceToHost));
    _local_to_global();
    cudaCheck(hipMemcpyAsync(device.joint_transforms, _joint_transforms.data(),
            _joint_transforms.size() * sizeof(float), hipMemcpyHostToDevice));

    // weights: (#verts, #joints)
    device::lbs<<<(model.verts.size() - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(
        device.joint_transforms, device.verts_shaped, model.device.weights.values,
        model.device.weights.inner, model.device.weights.outer,
        device.verts,
        model.n_joints(), model.n_verts());
}

// Instantiation
template class Body<model_config::SMPL>;
template class Body<model_config::SMPL_v1>;
template class Body<model_config::SMPLH>;
template class Body<model_config::SMPLX>;
template class Body<model_config::SMPLXpca>;
template class Body<model_config::SMPLX_v1>;
template class Body<model_config::SMPLXpca_v1>;

}  // namespace smplx
